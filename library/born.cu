#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "gpuconst.cuh"
#include "btree.cuh"

using namespace std;

#include "cudaKernels.cu"

void test_kernel_add_sourceArray(float *d_reflectivity, geometry param, dim3 grid, dim3 block)
{
    float *d_zeros;
    CHECK(hipMalloc((void **)&d_zeros, param.nbytes))       /* wavefield at t-2 */
    CHECK(hipMemset(d_zeros, 0, param.nbytes))
    kernel_add_sourceArray<<<grid,block>>>(d_zeros, d_reflectivity);

    float *h_zeros = new float[param.nbxy];
    CHECK(hipMemcpy(h_zeros, d_zeros, param.nbytes, hipMemcpyDeviceToHost));

    FILE *f_test = fopen("test_kernel_add_sourceArray", "w");

    fwrite(h_zeros, sizeof(float), param.nbxy, f_test);
    fclose(f_test);
}

//void test_kernel_applySourceArray(float dt, float *d_reflectivity, float *d_pField, float *d_vel, float *d_q)

void born(geometry param, velocity h_model, source h_wavelet, float *h_tapermask, seismicData h_seisData, sf_file Fonly_directWave, sf_file Fdata_directWave, sf_file Fdata, bool snaps)
{
    //hipStream_t stream1;
    //hipStreamCreate(&stream1);

    float dt2 = (h_wavelet.timeStep * h_wavelet.timeStep);
    float one_dx2 = float(1) / (param.modelDx * param.modelDx);
    float one_dy2 = float(1) / (param.modelDy * param.modelDy);
    //float dt2dx2 =
        //(h_wavelet.timeStep * h_wavelet.timeStep) / (param.modelDx * param.modelDx);   [> const for fd stencil <]
    size_t dbytes = param.nReceptors * h_wavelet.timeSamplesNt * sizeof(float);
    size_t tbytes = h_wavelet.timeSamplesNt * sizeof(float);

    //int bufferSize = min(param.nShots, 20);
    int bufferSize = min(param.nShots, 7);

    // Allocate memory on device
    printf("Allocate and copy memory on the device...\n");
    float *d_u1, *d_u2, *d_q1, *d_q2, *d_vp, *d_wavelet, *d_tapermask, *d_data, *d_directwave, *d_reflectivity;
    float *d_buffer, *d_lap;
    CHECK(hipMalloc((void **)&d_u1, param.nbytes))       /* wavefield at t-2 */
    CHECK(hipMalloc((void **)&d_u2, param.nbytes))       /* wavefield at t-1 */
    CHECK(hipMalloc((void **)&d_q1, param.nbytes))       /* wavefield at t-2 */
    CHECK(hipMalloc((void **)&d_q2, param.nbytes))       /* wavefield at t-1 */
    CHECK(hipMalloc((void **)&d_vp, param.nbytes))       /* velocity model */
    CHECK(hipMalloc((void **)&d_lap, param.nbytes))       /* velocity model */
    CHECK(hipMalloc((void **)&d_wavelet, tbytes)); /* source term for each time step */
    CHECK(hipMalloc((void **)&d_tapermask, param.nbytes));
    CHECK(hipMalloc((void **)&d_reflectivity, param.nxy * sizeof(float)));
    CHECK(hipMalloc((void **)&d_data, dbytes));
    CHECK(hipMalloc((void **)&d_directwave, dbytes));
    CHECK(hipMalloc((void **)&d_buffer, dbytes * bufferSize));

    // Fill allocated memory with a value
    CHECK(hipMemset(d_u1, 0, param.nbytes))
    CHECK(hipMemset(d_u2, 0, param.nbytes))
    CHECK(hipMemset(d_data, 0, dbytes))

    // Copy arrays from host to device
    CHECK(hipMemcpy(d_vp, h_model.extVelField, param.nbytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_tapermask, h_tapermask, param.nbytes, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_reflectivity, h_model.reflecitivy, param.nxy * sizeof(float), hipMemcpyHostToDevice));
    CHECK(hipMemcpy(d_wavelet, h_wavelet.timeSeries, tbytes, hipMemcpyHostToDevice));

    // Copy constants to device constant memory
    float coef[] = {a0, a1, a2, a3, a4};
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_coef), coef, 5 * sizeof(float)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_nx), &param.modelNxBorder, sizeof(int)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_ny), &param.modelNyBorder, sizeof(int)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_nr), &param.nReceptors, sizeof(int)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_nxy), &param.nbxy, sizeof(int)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_nb), &param.taperBorder, sizeof(int)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_nt), &h_wavelet.timeSamplesNt, sizeof(int)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_dt2), &dt2, sizeof(float)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_one_dx2), &one_dx2, sizeof(float)));
    CHECK(hipMemcpyToSymbol(HIP_SYMBOL(c_one_dy2), &one_dy2, sizeof(float)));
    printf("\t%f MB\n", (4 * param.nbytes + tbytes)/1024/1024);
    printf("OK\n");

    // Print out specs of the main GPU
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, 0));
    printf("GPU0:\t%s\t%d.%d:\n", deviceProp.name, deviceProp.major, deviceProp.minor);
    printf("\t%lu GB:\t total Global memory (gmem)\n", deviceProp.totalGlobalMem / 1024 / 1024 / 1000);
    printf("\t%lu MB:\t total Constant memory (cmem)\n", deviceProp.totalConstMem / 1024);
    printf("\t%lu MB:\t total Shared memory per block (smem)\n", deviceProp.sharedMemPerBlock / 1024);
    printf("\t%d:\t total threads per block\n", deviceProp.maxThreadsPerBlock);
    printf("\t%d:\t total registers per block\n", deviceProp.regsPerBlock);
    printf("\t%d:\t warp size\n", deviceProp.warpSize);
    printf("\t%d x %d x %d:\t max dims of block\n", deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
    printf("\t%d x %d x %d:\t max dims of grid\n", deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
    CHECK(hipSetDevice(0));

    // Print out CUDA domain partitioning info
    //printf("CUDA:\n");
    //printf("\t%i x %i\t:block dim\n", BDIMY, BDIMX);
    //printf("\t%i x %i\t:shared dim\n", SDIMY, SDIMX);
    //printf("CFL:\n");
    //printf("\t%f\n", _vp * h_wavelet.timeStep / dx);

    // Setup CUDA run
    dim3 block(BDIMX, BDIMY);
    dim3 grid((param.modelNxBorder + block.x - 1) / block.x, (param.modelNyBorder + block.y - 1) / block.y);

    //test_kernel_add_sourceArray(d_reflectivity, param, grid, block);


    int seismicSize = param.nReceptors * h_wavelet.timeSamplesNt;
    int step = 0;

    // MAIN LOOP
    for(int shot=0; shot<param.nShots; shot++){
        cerr<<"\nShot "<<shot<<" param.firstReceptorPos = "<<param.firstReceptorPos<<", param.srcPosX = "<<param.srcPosX<<", param.srcPosY = "<<param.srcPosY<<
            ", param.incShots = "<<param.incShots<<"\n"<<endl;

        CHECK(hipMemset(d_u1, 0, param.nbytes))
        CHECK(hipMemset(d_u2, 0, param.nbytes))
        CHECK(hipMemset(d_q1, 0, param.nbytes))
        CHECK(hipMemset(d_q2, 0, param.nbytes))


        float *d_u3, *d_q3;
        printf("Time loop...\n");
        for (int it = 0; it < h_wavelet.timeSamplesNt; it++)
        {
            taper_gpu<<<grid,block>>>(d_tapermask, d_u1);
            //testegoxtoso<<<grid,block>>>(d_u1);
            taper_gpu<<<grid,block>>>(d_tapermask, d_u2);
            //if(it == 0) saveSnapshotIstep(it, d_u1, param.modelNxBorder, param.modelNyBorder, "u1", shot);

            // These kernels are in the same stream so they will be executed one by one
            //kernel_2dfd<<<grid, block>>>(d_u1, d_u2, d_vp);
            kernel_2dfd_ver2<<<grid, block>>>(d_lap, d_u1, d_u2, d_vp);
            kernel_add_wavelet<<<grid, block>>>(d_u2, d_wavelet, it, param.srcPosX, param.srcPosY);

            taper_gpu<<<grid,block>>>(d_tapermask, d_q1);
            taper_gpu<<<grid,block>>>(d_tapermask, d_q2);

            // These kernels are in the same stream so they will be executed one by one
            kernel_2dfd<<<grid, block>>>(d_q1, d_q2, d_vp);
            //kernel_applySourceArray<<<grid, block>>>(h_wavelet.timeStep, d_reflectivity, d_u2, d_vp, d_q1);
            kernel_applySourceArray_ver2<<<grid, block>>>(h_wavelet.timeStep, d_reflectivity, d_lap, d_vp, d_q1);

            receptors<<<(param.nReceptors + 32) / 32, 32>>>(it, param.nReceptors, param.firstReceptorPos, d_q1, d_data);

            // Save snapshot every h_wavelet.snapStep iterations
            if ((it % h_wavelet.snapStep == 0) && snaps == true)
            {
                printf("%i/%i\n", it+1, h_wavelet.timeSamplesNt);
                saveSnapshotIstep(it, d_u1, param.modelNxBorder, param.modelNyBorder, "u1", shot);
                saveSnapshotIstep(it, d_q1, param.modelNxBorder, param.modelNyBorder, "q1", shot);
            }

            // Exchange time steps
            d_u3 = d_u1;
            d_u1 = d_u2;
            d_u2 = d_u3;

            d_q3 = d_q1;
            d_q1 = d_q2;
            d_q2 = d_q3;
        }

        //CHECK(hipMemcpyAsync(h_seisData.seismogram, d_data, dbytes, hipMemcpyDeviceToHost, stream1));
        //CHECK(hipMemcpy(h_seisData.seismogram, d_data, dbytes, hipMemcpyDeviceToHost));

        sf_warning("in_step=%d",step);
        if (step == bufferSize)
        {
            sf_warning("shot - bufferSize + 1 = %d",shot - bufferSize);
            CHECK(hipMemcpy(&h_seisData.seismogram[(shot - bufferSize) * seismicSize], d_buffer, dbytes * bufferSize, hipMemcpyDeviceToHost));
            step = 0;
        }
        CHECK(hipMemcpy(&d_buffer[step * seismicSize], d_data, dbytes, hipMemcpyDeviceToDevice));
        step += 1;



        param.firstReceptorPos += param.incRec;
        param.srcPosX += param.incShots;
    }


    sf_warning("out_step=%d",step);
    if(step < bufferSize)
    {
        CHECK(hipMemcpy(&h_seisData.seismogram[(param.nShots - step) * seismicSize], d_buffer, dbytes * step, hipMemcpyDeviceToHost));
    }
    else if (param.nShots == bufferSize)
    {
        CHECK(hipMemcpy(h_seisData.seismogram, d_buffer, dbytes * param.nShots, hipMemcpyDeviceToHost));
    }

    sf_floatwrite(h_seisData.seismogram, param.nReceptors * h_wavelet.timeSamplesNt * param.nShots, Fdata);

    printf("OK\n");

    CHECK(hipGetLastError());


    CHECK(hipFree(d_q1));
    CHECK(hipFree(d_q2));
    CHECK(hipFree(d_u1));
    CHECK(hipFree(d_lap));
    CHECK(hipFree(d_u2));
    CHECK(hipFree(d_tapermask));
    CHECK(hipFree(d_reflectivity));
    CHECK(hipFree(d_data));
    CHECK(hipFree(d_directwave));
    CHECK(hipFree(d_vp));
    CHECK(hipFree(d_wavelet));
    CHECK(hipFree(d_buffer));
    printf("OK saigo\n");
    CHECK(hipDeviceReset());
    //hipStreamDestroy(&stream1);
}
