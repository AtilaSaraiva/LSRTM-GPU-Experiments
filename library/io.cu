#include "hip/hip_runtime.h"
#include <iostream>
#include "btree.cuh"

using namespace std;


void dummyVelField(int nxb, int nyb, int nb, float *h_vpe, float *h_dvpe)
{
    for (int i = 0; i < nyb; i++){
        for (int j = 0; j < nxb; j++){
            h_dvpe[j * nyb + i]  = h_vpe[j * nyb + nb];
        }
    }
}

void expand(int nb, int nyb, int nxb, int nz, int nx, float *a, float *b)
/*< expand domain of 'a' to 'b':  a, size=nz*nx; b, size=nyb*nxb;  >*/
{
    int iz,ix;
    for     (ix=0;ix<nx;ix++) {
        for (iz=0;iz<nz;iz++) {
            b[(nb+ix)*nyb+(nb+iz)] = a[ix*nz+iz];
        }
    }
    for     (ix=0; ix<nxb; ix++) {
        for (iz=0; iz<nb; iz++)         b[ix*nyb+iz] = b[ix*nyb+nb];//top
        for (iz=nz+nb; iz<nyb; iz++) b[ix*nyb+iz] = b[ix*nyb+nb+nz-1];//bottom
    }
    for (iz=0; iz<nyb; iz++){
        for(ix=0; ix<nb; ix++)  b[ix*nyb+iz] = b[nb*nyb+iz];//left
        for(ix=nb+nx; ix<nxb; ix++)     b[ix*nyb+iz] = b[(nb+nx-1)*nyb+iz];//right
    }
}

void abc_coef (int nb, float *abc)
{
    for(int i=0; i<nb; i++){
        abc[i] = exp (-pow(0.002 * (nb - i + 1),2.0));
    }
}

void taper (int nx, int ny, int nb, float *abc, float *campo)
{
    int nxb = nx + 2 * nb;
    int nyb = ny + 2 * nb;
    for(int j=0; j<nxb; j++){
        for(int i=0; i<nb; i++){
            campo[j * nyb + i] *= abc[i];
            campo[j * nyb + (nb + ny + i)] *= abc[nb - i - 1];
        }
    }
    for(int i=0; i<nyb; i++){
        for(int j=0; j<nb; j++){
            campo[j * nyb + i] *= abc[j];
            campo[(nb + nx + j) * nyb + i] *= abc[nb - j - 1];
        }
    }
}

sf_file createFile3D (const char *name, int dimensions[3], float spacings[3], int origins[3])
{
    sf_file Fdata = NULL;
    Fdata = sf_output(name);
    char key_n[6],key_d[6],key_o[6];
    for (int i = 0; i < 3; i++){
        sprintf(key_n,"n%i",i+1);
        sprintf(key_d,"d%i",i+1);
        sprintf(key_o,"o%i",i+1);
        sf_putint(Fdata,key_n,dimensions[i]);
        sf_putfloat(Fdata,key_d,spacings[i]);
        sf_putint(Fdata,key_o,origins[i]);
    }

    return Fdata;
}

geometry getParameters(sf_file FvelModel, sf_file Fshots)
{
    geometry param;
    sf_histint(Fshots,"n2",&param.nReceptors);
    sf_histint(Fshots,"sybeg",&param.srcPosY);
    sf_histint(Fshots,"sxbeg",&param.srcPosX);
    sf_histint(Fshots,"gxbeg",&param.firstReceptorPos);
    sf_histint(Fshots,"n3",&param.nShots);
    sf_histint(Fshots,"incShots",&param.incShots);
    sf_histint(Fshots,"incRec",&param.incRec);
    sf_histint(FvelModel, "n1",&param.modelNy);
    sf_histint(FvelModel, "n2", &param.modelNx);
    sf_histfloat(FvelModel, "d1",&param.modelDy);
    sf_histfloat(FvelModel, "d2", &param.modelDx);
    param.lastReceptorPos = param.firstReceptorPos + param.nReceptors;
    param.taperBorder = 0.3 * param.modelNx;
    param.nxy = param.modelNx * param.modelNy;
    param.modelNxBorder = param.modelNx + 2 * param.taperBorder;
    param.modelNyBorder = param.modelNy + 2 * param.taperBorder;
    param.nbxy = param.modelNxBorder * param.modelNyBorder;
    param.nbytes = param.nbxy * sizeof(float); // bytes to store modelNxBorder * modelNyBorder
    return param;
}

geometry getParameters(sf_file FvelModel)
{
    geometry param;
    sf_getint("nr",&param.nReceptors);
    sf_getint("isrc",&param.srcPosY);
    sf_getint("jsrc",&param.srcPosX);
    sf_getint("gxbeg",&param.firstReceptorPos);
    sf_getint("nshots",&param.nShots);
    sf_getint("incShots",&param.incShots);
    sf_getint("incRec",&param.incRec);
    sf_histint(FvelModel, "n1",&param.modelNy);
    sf_histint(FvelModel, "n2", &param.modelNx);
    sf_histfloat(FvelModel, "d1",&param.modelDy);
    sf_histfloat(FvelModel, "d2", &param.modelDx);
    param.lastReceptorPos = param.firstReceptorPos + param.nReceptors;
    param.taperBorder = 0.3 * param.modelNx;
    param.nxy = param.modelNx * param.modelNy;
    param.modelNxBorder = param.modelNx + 2 * param.taperBorder;
    param.modelNyBorder = param.modelNy + 2 * param.taperBorder;
    param.nbxy = param.modelNxBorder * param.modelNyBorder;
    param.nbytes = param.nbxy * sizeof(float); // bytes to store modelNxBorder * modelNyBorder
    return param;
}

velocity getVelFields(sf_file FvelModel, geometry param)
{
    velocity h_model;

    h_model.velField = new float[param.nxy];
    sf_floatread(h_model.velField, param.nxy, FvelModel);

    h_model.extVelField = new float[param.nbxy];
    memset(h_model.extVelField,0,param.nbytes);
    expand(param.taperBorder, param.modelNyBorder, param.modelNxBorder, param.modelNy, param.modelNx, h_model.velField, h_model.extVelField);

    h_model.maxVel = h_model.velField[0];
    for(int i=1; i < param.nxy; i++){
        if(h_model.velField[i] > h_model.maxVel){
            h_model.maxVel = h_model.velField[i];
        }
    }

    h_model.firstLayerVelField = new float[param.nbxy];
    dummyVelField(param.modelNxBorder, param.modelNyBorder, param.taperBorder, h_model.extVelField, h_model.firstLayerVelField);

    printf("MODEL:\n");
    printf("\t%i x %i\t:param.modelNy x param.modelNx\n", param.modelNy, param.modelNx);
    printf("\t%f\t:param.modelDx\n", param.modelDx);
    printf("\t%f\t:h_model.velField[0]\n", h_model.velField[0]);
    return h_model;
}

velocity getVelFields(sf_file FvelModel, sf_file Freflectivity, geometry param)
{
    velocity h_model;

    h_model.velField = new float[param.nxy];
    sf_floatread(h_model.velField, param.nxy, FvelModel);

    h_model.reflecitivy = new float[param.nxy];
    sf_floatread(h_model.reflecitivy, param.nxy, Freflectivity);

    h_model.extVelField = new float[param.nbxy];
    memset(h_model.extVelField,0,param.nbytes);
    expand(param.taperBorder, param.modelNyBorder, param.modelNxBorder, param.modelNy, param.modelNx, h_model.velField, h_model.extVelField);

    h_model.maxVel = h_model.velField[0];
    for(int i=1; i < param.nxy; i++){
        if(h_model.velField[i] > h_model.maxVel){
            h_model.maxVel = h_model.velField[i];
        }
    }

    h_model.firstLayerVelField = new float[param.nbxy];
    dummyVelField(param.modelNxBorder, param.modelNyBorder, param.taperBorder, h_model.extVelField, h_model.firstLayerVelField);

    printf("MODEL:\n");
    printf("\t%i x %i\t:param.modelNy x param.modelNx\n", param.modelNy, param.modelNx);
    printf("\t%f\t:param.modelDx\n", param.modelDx);
    printf("\t%f\t:h_model.velField[0]\n", h_model.velField[0]);
    return h_model;
}

float* tapermask(geometry param)
{
    float *h_abc = new float[param.taperBorder];
    float *h_tapermask = new float[param.nbxy];
    for(int i=0; i < param.nbxy; i++){
        h_tapermask[i] = 1;
    }
    abc_coef(param.taperBorder, h_abc);
    taper(param.modelNx, param.modelNy, param.taperBorder, h_abc, h_tapermask);
    delete[] h_abc;
    return h_tapermask;
}


seismicData allocHostSeisData(geometry param, int nt)
{
    seismicData h_seisData;
    h_seisData.seismogram = new float[param.nReceptors * nt];
    h_seisData.directWaveOnly = new float[param.nReceptors * nt];
    return h_seisData;
}

seismicData allocHostSeisData(geometry param, sf_file Fshots)
{
    seismicData h_seisData;
    sf_histfloat(Fshots,"d1",&h_seisData.timeStep);
    sf_histint(Fshots,"n1",&h_seisData.timeSamplesNt);
    h_seisData.seismogram = new float[param.nShots * param.nReceptors * h_seisData.timeSamplesNt];
    sf_floatread(h_seisData.seismogram, param.nShots * param.nReceptors * h_seisData.timeSamplesNt, Fshots);
    return h_seisData;
}


float* fillSrc(geometry param, velocity h_model, seismicData h_seisData)
{
    float* wavelet;

    float f0 = 10.0;                    // source dominawavelet.timeSamplesNt frequency, Hz <]
    float t0 = 1.2 / f0;                // source padding to move wavelet from left of zero <]

    float tbytes = h_seisData.timeSamplesNt * sizeof(float);
    float* time = (float *)malloc(tbytes);
    wavelet = (float *)malloc(tbytes);

    // Fill source waveform vector
    float a = PI * PI * f0 * f0;            // const for wavelet <]
    float dt2dx2 = (h_seisData.timeStep * h_seisData.timeStep) / (param.modelDx * param.modelDx);   // const for fd stencil <]
    for (int it = 0; it < h_seisData.timeSamplesNt; it++)
    {
        time[it] = it * h_seisData.timeStep;
        // Ricker wavelet (Mexican hat), second derivative of Gaussian
        wavelet[it] = 1e10 * (1.0 - 2.0 * a * pow(time[it] - t0, 2)) * exp(-a * pow(time[it] - t0, 2));
        wavelet[it] *= dt2dx2;
    }
    delete[] time;
    //printf("TIME STEPPING:\n");
    //printf("\t%e\t:h_seisData.timeStep\n", h_seisData.timeStep);
    //printf("\t%i\t:h_seisData.timeSamplesNt\n", h_seisData.timeSamplesNt);
    return wavelet;
}

source fillSrc(geometry param, velocity h_model)
{
    source wavelet;
    wavelet.totalTime = 3;               /* total time of wave propagation, sec */
    float one_dx2 = float(1) / (param.modelDx * param.modelDx);
    float one_dy2 = float(1) / (param.modelDy * param.modelDy);
    wavelet.timeStep = 0.5 / (h_model.maxVel * sqrt(one_dx2 + one_dy2)) ;         /* time step assuming constant vp, sec */
    wavelet.timeSamplesNt = round(wavelet.totalTime / wavelet.timeStep);    // number of time steps
    wavelet.snapStep = round(0.1 * wavelet.timeSamplesNt);   /* save snapshot every ... steps */

    float f0 = 10.0;                    /* source dominawavelet.timeSamplesNt frequency, Hz */
    float t0 = 1.2 / f0;                /* source padding to move wavelet from left of zero */

    float tbytes = wavelet.timeSamplesNt * sizeof(float);
    float* time = (float *)malloc(tbytes);
    wavelet.timeSeries = (float *)malloc(tbytes);

    // Fill source waveform vector
    float a = PI * PI * f0 * f0;            /* const for wavelet */
    float dt2dx2 = (wavelet.timeStep * wavelet.timeStep) / (param.modelDx * param.modelDx);   /* const for fd stencil */
    for (int it = 0; it < wavelet.timeSamplesNt; it++)
    {
        time[it] = it * wavelet.timeStep;
        // Ricker wavelet (Mexican hat), second derivative of Gaussian
        wavelet.timeSeries[it] = 1e10 * (1.0 - 2.0 * a * pow(time[it] - t0, 2)) * exp(-a * pow(time[it] - t0, 2));
        wavelet.timeSeries[it] *= dt2dx2;
    }
    delete[] time;
    printf("TIME STEPPING:\n");
    printf("\t%e\t:h_wavelet.totalTime\n", wavelet.totalTime);
    printf("\t%e\t:h_wavelet.timeStep\n", wavelet.timeStep);
    printf("\t%i\t:h_wavelet.timeSamplesNt\n", wavelet.timeSamplesNt);
    return wavelet;
}

void test_getParameters (geometry param, seismicData h_seisData)
{
    cerr<<"param.incShots: "<<param.incShots<<endl;
    cerr<<"param.incShots: "<<param.incShots<<endl;
    cerr<<"param.modelDims nx = "<<param.modelNx<<" ny = "<<param.modelNy<<endl;
    cerr<<"param.modelDx = "<<param.modelDx<<" param.modelDy = "<<param.modelDy<<endl;
    cerr<<"param.taperBorder = "<<param.taperBorder<<endl;
    cerr<<"param.nShots "<<param.nShots<<endl;
    cerr<<"param.nReceptors "<<param.nReceptors<<endl;
    cerr<<"param.firstReceptorPos "<<param.firstReceptorPos<<endl;
    cerr<<"param.lastReceptorPos "<<param.lastReceptorPos<<endl;
    cerr<<"h_seisData.timeSamplesNt "<<h_seisData.timeSamplesNt<<endl;
    cerr<<"h_seisData.timeStep "<<h_seisData.timeStep<<endl;
}
void test_getParameters (geometry param, source wavelet)
{
    cerr<<"param.incShots: "<<param.incShots<<endl;
    cerr<<"param.modelDims nx = "<<param.modelNx<<" ny = "<<param.modelNy<<endl;
    cerr<<"param.modelDx = "<<param.modelDx<<" param.modelDy = "<<param.modelDy<<endl;
    cerr<<"param.taperBorder = "<<param.taperBorder<<endl;
    cerr<<"param.nShots "<<param.nShots<<endl;
    cerr<<"param.nReceptors "<<param.nReceptors<<endl;
    cerr<<"param.firstReceptorPos "<<param.firstReceptorPos<<endl;
    cerr<<"param.lastReceptorPos "<<param.lastReceptorPos<<endl;
    cerr<<"wavelet.timeSamplesNt "<<wavelet.timeSamplesNt<<endl;
    cerr<<"wavelet.timeStep "<<wavelet.timeStep<<endl;
}
