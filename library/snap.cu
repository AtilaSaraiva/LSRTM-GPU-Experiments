#include "btree.cuh"
#include <stdio.h>

// Save snapshot as a binary, filename snap/snap_tag_it_ny_nx
void saveSnapshotIstep(int it, float *data, int nx, int ny, const char *tag, int shot)
{
    /*
    it      :timestep id
    data    :pointer to an array in device memory
    nx, ny  :model dimensions
    tag     :user-defined file identifier
    */

    // Array to store wavefield
    unsigned int isize = nx * ny * sizeof(float);
    float *iwave = (float *)malloc(isize);
    CHECK(hipMemcpy(iwave, data, isize, hipMemcpyDeviceToHost));

    char fname[32];
    sprintf(fname, "snap/snap_%s_s%i_%i_%i_%i", tag, shot, it, ny, nx);

    FILE *fp_snap = fopen(fname, "w");

    fwrite(iwave, sizeof(float), nx * ny, fp_snap);
    printf("\tSave...%s: nx = %i ny = %i it = %i tag = %s\n", fname, nx, ny, it, tag);
    fflush(stdout);
    fclose(fp_snap);

    free(iwave);
    return;
}
